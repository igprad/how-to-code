#include <exception>
#include <fstream>
#include <iostream>
#include <sstream>
#include <string>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


#define DEFAULT_MAX_ROWS 1000000

__device__ double sum(double first_val, double second_val, double third_val) {
  return first_val + second_val + third_val;
}

__global__ void sum_csv_data_with_cuda(double *result, double *first_col,
                                       double *second_col, double *third_col,
                                       int max) {
  int offset_x = threadIdx.x;
  int offset_y = blockIdx.x;
  int offset = offset_y + offset_x * max;
  if (offset < DEFAULT_MAX_ROWS)
    result[offset] =
        sum(first_col[offset], second_col[offset], third_col[offset]);
}

int main(int argc, char **argv) {
  // parse args
  int thread_in, block_in;
  try {
    thread_in = std::stoi(argv[1]);
    block_in = std::stoi(argv[2]);
  } catch (const std::exception &e) {
    std::cerr << "Invalid arguments. Please only pass the correct thread and "
                 "block. e.g. 1000 1000";
    return 0;
  }

  std::ifstream csv("dataset.csv");
  double *first_value = (double *)malloc(sizeof(double) * DEFAULT_MAX_ROWS);
  double *second_value = (double *)malloc(sizeof(double) * DEFAULT_MAX_ROWS);
  double *third_value = (double *)malloc(sizeof(double) * DEFAULT_MAX_ROWS);
  double *result = (double *)malloc(sizeof(double) * DEFAULT_MAX_ROWS);

  // Getting the value from dataset
  std::string value;
  int idx = 0;
  bool csv_title_skipped = false;
  while (std::getline(csv, value)) {
    if (!csv_title_skipped) {
      csv_title_skipped = true;
      continue;
    }
    std::string number;
    std::stringstream lineToInputBuffered(value);
    std::string::size_type sz;
    int colIdx = 0;
    while (std::getline(lineToInputBuffered, number, ',')) {
      try {
        if (colIdx == 1)
          first_value[idx] = std::stod(number, &sz);
        if (colIdx == 2)
          second_value[idx] = std::stod(number, &sz);
        if (colIdx == 3)
          third_value[idx] = std::stod(number, &sz);
      } catch (const std::exception &e) {
        std::cerr << "error: " << e.what() << std::endl;
      }
      colIdx++;
    }
    idx++;
  }

  /*
   * Will calculate the time spent to sum all values for each rows.
   * This section will be quite different with `sum.cpp`,
   * since will use cuda api to do the parallel tasks.
   * */
  double *first_dev = (double *)malloc(sizeof(double) * DEFAULT_MAX_ROWS);
  hipMalloc(&first_dev, sizeof(double) * DEFAULT_MAX_ROWS);
  hipMemcpy(first_dev, first_value, sizeof(double) * DEFAULT_MAX_ROWS,
             hipMemcpyHostToDevice);

  double *second_dev = (double *)malloc(sizeof(double) * DEFAULT_MAX_ROWS);
  hipMalloc(&second_dev, sizeof(double) * DEFAULT_MAX_ROWS);
  hipMemcpy(second_dev, second_value, sizeof(double) * DEFAULT_MAX_ROWS,
             hipMemcpyHostToDevice);

  double *third_dev = (double *)malloc(sizeof(double) * DEFAULT_MAX_ROWS);
  hipMalloc(&third_dev, sizeof(double) * DEFAULT_MAX_ROWS);
  hipMemcpy(third_dev, third_value, sizeof(double) * DEFAULT_MAX_ROWS,
             hipMemcpyHostToDevice);

  double *result_dev = (double *)malloc(sizeof(double) * DEFAULT_MAX_ROWS);
  hipMalloc(&result_dev, sizeof(double) * DEFAULT_MAX_ROWS);

  hipEvent_t start, stop;
  float time = 0;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start, 0);
  sum_csv_data_with_cuda<<<thread_in, block_in>>>(
      result_dev, first_dev, second_dev, third_dev, block_in);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);

  hipMemcpy(result, result_dev, sizeof(double) * DEFAULT_MAX_ROWS,
             hipMemcpyDeviceToHost);

  hipFree(first_dev);
  hipFree(second_dev);
  hipFree(third_dev);
  hipFree(result_dev);
  hipFree(&start);
  hipFree(&stop);
  /*
   * end section
   */

  double total = 0;
  for (int i = 0; i < DEFAULT_MAX_ROWS; i++) {
    total += result[i];
  }
  std::cout << "Total sum of all rows = " << total << std::endl;
  std::cout << "With elapsed time while using cuda threads (ms) = " << time
            << std::endl;

  free(first_value);
  free(second_value);
  free(third_value);
  free(result);
}
