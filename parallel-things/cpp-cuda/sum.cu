#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <chrono>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


#define MAX 1000000


__device__ double sum(double first_val, double second_val, double third_val) {
    return first_val + second_val + third_val;
}

__global__ void sum_csv_data_with_cuda(double *result, double *first_row, double *second_row, double *third_row, int max) {
    int offset_x = threadIdx.x;
    int offset_y = blockIdx.x;
    int offset = offset_y + offset_x * max;
    if (offset < MAX) result[offset] = sum(first_row[offset], second_row[offset], third_row[offset]);
}

int main() {
    std::ifstream csv ("dataset.csv");
    double* first_value = (double*) malloc(sizeof(double) * MAX);
    double* second_value = (double*) malloc(sizeof(double) * MAX);
    double* third_value = (double*) malloc(sizeof(double) * MAX);
    double* result = (double*)malloc(sizeof(double) * MAX);

    // Getting the value from dataset
    std::string value;
    int idx = 0;
    bool csv_title_skipped = false;
    while (std::getline(csv, value)) {
        if (!csv_title_skipped) {
            csv_title_skipped = true;
            continue;
        }
        std::string number;
        std::stringstream lineToInputBuffered(value);
        std::string::size_type sz;
        int rowIdx = 0;
        while (std::getline(lineToInputBuffered, number, ',')) {
            try {
                if (rowIdx == 1)
                   first_value[idx] = std::stod(number, &sz);
                if (rowIdx == 2)
                   second_value[idx] = std::stod(number, &sz);
                if (rowIdx == 3)
                   third_value[idx] = std::stod(number, &sz);
            } catch (const std::exception& e) {
                std::cerr << "error: "<< e.what() << std::endl;
            }
            rowIdx++;
        }
        idx++;
    }

    /*
     * Will calculate the time spent to sum all values for each rows.
     * This section will be quite different with `sum.cpp`,
     * since will use cuda api to do the parallel tasks.
     * */
    double* first_dev = (double*) malloc(sizeof(double) * MAX);
    hipMalloc(&first_dev, sizeof(double) * MAX);
    hipMemcpy(first_dev, first_value, sizeof(double) * MAX, hipMemcpyHostToDevice);

    double* second_dev = (double*) malloc(sizeof(double) * MAX);
    hipMalloc(&second_dev, sizeof(double) * MAX);
    hipMemcpy(second_dev, second_value, sizeof(double) * MAX, hipMemcpyHostToDevice);

    double* third_dev = (double*) malloc(sizeof(double) * MAX);
    hipMalloc(&third_dev, sizeof(double) * MAX);
    hipMemcpy(third_dev, third_value, sizeof(double) * MAX, hipMemcpyHostToDevice);
    
    double* result_dev = (double*) malloc(sizeof(double) * MAX);
    hipMalloc(&result_dev, sizeof(double) * MAX);

    hipEvent_t start, stop;
    float time = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    sum_csv_data_with_cuda<<<1000, 1000>>>(result_dev, first_dev, second_dev, third_dev, 1000); 
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    hipMemcpy(result, result_dev, sizeof(double) * MAX, hipMemcpyDeviceToHost);

    hipFree(first_dev);
    hipFree(second_dev);
    hipFree(third_dev);
    hipFree(result_dev);
    hipFree(&start);
    hipFree(&stop);
    /*
     * end section
     */

    double total = 0;
    for(int i=0; i < MAX; i++) {
        total+=result[i];
    }
    std::cout<<"Total sum of all rows = " << total << std::endl;
    std::cout<<"With elapsed time while using cuda threads (ms) = " << time << std::endl;

    free(first_value);
    free(second_value);
    free(third_value);
    free(result);
    return 0;
}
