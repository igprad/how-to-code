#include <exception>
#include <fstream>
#include <iostream>
#include <sstream>
#include <string>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


#define DEFAULT_MAX_ROWS 1000000

__device__ double sum(double first_val, double second_val, double third_val) {
  return first_val + second_val + third_val;
}

__global__ void sum_csv_data_with_cuda(double *result, double *first_col,
                                       double *second_col, double *third_col) {
  int offset_x = threadIdx.x;
  int offset_y = blockIdx.x;
  int offset = offset_x + offset_y * 1000;
  if (offset < DEFAULT_MAX_ROWS)
    result[offset] =
        sum(first_col[offset], second_col[offset], third_col[offset]);
}

int main(int argc, char **argv) {

  std::ifstream csv("dataset.csv");
  double *first_value = (double *)malloc(sizeof(double) * DEFAULT_MAX_ROWS);
  double *second_value = (double *)malloc(sizeof(double) * DEFAULT_MAX_ROWS);
  double *third_value = (double *)malloc(sizeof(double) * DEFAULT_MAX_ROWS);
  double *result = (double *)malloc(sizeof(double) * DEFAULT_MAX_ROWS);

  // Getting the value from dataset
  std::string value;
  int idx = 0;
  bool csv_title_skipped = false;
  while (std::getline(csv, value)) {
    if (!csv_title_skipped) {
      csv_title_skipped = true;
      continue;
    }
    std::string number;
    std::stringstream lineToInputBuffered(value);
    std::string::size_type sz;
    int colIdx = 0;
    while (std::getline(lineToInputBuffered, number, ',')) {
      try {
        if (colIdx == 1)
          first_value[idx] = std::stod(number, &sz);
        if (colIdx == 2)
          second_value[idx] = std::stod(number, &sz);
        if (colIdx == 3)
          third_value[idx] = std::stod(number, &sz);
      } catch (const std::exception &e) {
        std::cerr << "error: " << e.what() << std::endl;
      }
      colIdx++;
    }
    idx++;
  }

  /*
   * Will calculate the time spent to sum all values for each rows.
   * This section will be quite different with `sum.cpp`,
   * since will use cuda api to do the parallel tasks.
   * */
  double *first_dev = (double *)malloc(sizeof(double) * DEFAULT_MAX_ROWS);
  hipMalloc(&first_dev, sizeof(double) * DEFAULT_MAX_ROWS);
  hipMemcpy(first_dev, first_value, sizeof(double) * DEFAULT_MAX_ROWS,
             hipMemcpyHostToDevice);

  double *second_dev = (double *)malloc(sizeof(double) * DEFAULT_MAX_ROWS);
  hipMalloc(&second_dev, sizeof(double) * DEFAULT_MAX_ROWS);
  hipMemcpy(second_dev, second_value, sizeof(double) * DEFAULT_MAX_ROWS,
             hipMemcpyHostToDevice);

  double *third_dev = (double *)malloc(sizeof(double) * DEFAULT_MAX_ROWS);
  hipMalloc(&third_dev, sizeof(double) * DEFAULT_MAX_ROWS);
  hipMemcpy(third_dev, third_value, sizeof(double) * DEFAULT_MAX_ROWS,
             hipMemcpyHostToDevice);

  double *result_dev = (double *)malloc(sizeof(double) * DEFAULT_MAX_ROWS);
  hipMalloc(&result_dev, sizeof(double) * DEFAULT_MAX_ROWS);

  hipEvent_t start, stop;
  float time = 0;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start, 0);
  sum_csv_data_with_cuda<<<1000, 1000>>>(result_dev, first_dev, second_dev,
                                         third_dev);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time, start, stop);

  hipMemcpy(result, result_dev, sizeof(double) * DEFAULT_MAX_ROWS,
             hipMemcpyDeviceToHost);

  hipFree(first_dev);
  hipFree(second_dev);
  hipFree(third_dev);
  hipFree(result_dev);
  hipFree(&start);
  hipFree(&stop);
  /*
   * end section
   */

  double total = 0;
  for (int i = 0; i < DEFAULT_MAX_ROWS; i++) {
    total += result[i];
  }
  std::cout << "Total sum of all rows = " << (int)total << std::endl;
  std::cout << "With elapsed time while using cuda threads (ms) = " << time
            << std::endl;

  free(first_value);
  free(second_value);
  free(third_value);
  free(result);
}
